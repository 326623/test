#include "hip/hip_runtime.h"
/* Copyright (C) 2018 New Joy - All Rights Reserved
 * You may use, distribute and modify this code under the
 * terms of the GPLv3
 *
 *
 * You should have received a copy of the GPLv3 license with
 * this file. If not, please visit https://www.gnu.org/licenses/gpl-3.0.en.html
 *
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * Author: yangqp5@outlook.com (New Joy)
 *
 */
#include <benchmark/benchmark.h>
#include <gtest/gtest.h>
#include <iostream>
#include <vector>
#include <memory>

#include <hip/hip_runtime.h>
#include "scan.cuh"

// note: this implementation does not disable this overload for array types
// template <typename T, typename... Args>
// std::unique_ptr<T> make_unique(Args&&... args) {
//   return std::unique_ptr<T>(new T(std::forward<Args>(args)...));
// }
// template <class T, class ...Args>
// typename std::enable_if
// <
//     !std::is_array<T>::value,
//     std::unique_ptr<T>
// >::type
// make_unique(Args&& ...args)
// {
//     return std::unique_ptr<T>(new T(std::forward<Args>(args)...));
// }

// template <class T>
// typename std::enable_if
// <
//     std::is_array<T>::value,
//     std::unique_ptr<T>
// >::type
// make_unique(std::size_t n)
// {
//     typedef typename std::remove_extent<T>::type RT;
//     return std::unique_ptr<T>(new RT[n]);
// }

// __global__ void generate_from(float* first, float* last, int start_from) {
//   int id = threadIdx.x + blockIdx.x * blockDim.x;
//   int num_elements = last - first;
//   for (int i = id; i < num_elements; i += blockDim.x * gridDim.x) {
//     first[i] = start_from + i;
//   }
// }

// // int main() {
// //   const int kN = 100;
// //   std::size_t size = N * sizeof(float);
// //   auto h_A = make_unique<float>(size);
// //   float& d_A;
// //   hipMalloc(&d_A, size);
// //   hipError_t error = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
// //   // if (error != hipSuccess) {
// //   // }
// //   inclusive_prefix_sum<16 * 16, 16 * 16>(
// //       d_A, d_A + size, 0, [](float a, float b) { return a + b; });
// // }
// TEST(TEST_SCAN, TEST1) {
//   const int kSize = 1000;
//   auto h_A = make_unique<float[]>(kSize);
//   // float* hh_A = new float[kSize];
//   float* d_A;
//   hipMalloc(&d_A, kSize * sizeof(float));
//   hipError_t error = hipMemcpy(d_A, h_A.get(), kSize * sizeof(float), hipMemcpyHostToDevice);
//   // hipEvent_t stop;
//   if (error != hipSuccess) {
//     std::exit(-1);
//   }
//   generate_from<<<16 * 16, 16 * 16>>>(d_A, d_A + kSize, 1);
//   // hipDeviceSynchronize();
//   inclusive_prefix_sum<<<16 * 16, 16 * 16>>>(
//       d_A, d_A + kSize);
//   error = hipMemcpy(h_A.get(), d_A, kSize * sizeof(float), hipMemcpyDeviceToHost);
//   hipDeviceSynchronize();

//   // checking correct
//   for (int i = 0; i < kSize; ++ i) {
//     EXPECT_EQ(h_A[i], (i+2) * (i+1) / 2);
//   }
//   hipFree(d_A);
//   // error = hipEventSynchronize(&stop);
// }
