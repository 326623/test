
#include <hip/hip_runtime.h>
__global__ void add(int *a, int *b, int *c) {
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int *a, int size) {
  int i = 0;
  for (i = 0; i < size; ++ i)
    a[i] = rand();
}

#define N 512
int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;

  int size = N * sizeof(int);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<<1, N>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}