#include <absl/strings/str_format.h>
#include <benchmark/benchmark.h>
#include <glog/logging.h>
#include <gtest/gtest.h>

#include <iostream>
#include <random>

#include <hiprand.h>

#include "cuda_macro.h"
#include "reduction.cuh"

class RandomArrayFixture : public benchmark::Fixture {
 public:
  void SetUp(const ::benchmark::State& state) {
    n = state.range(0);
    CUDA_CALL(hipMalloc(&device_array, n * sizeof(float)));
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
  }

  void TearDown(const ::benchmark::State& state) {
    CUDA_CALL(hipFree(device_array));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
  }

 protected:
  float* device_array;
  int n;
  hipEvent_t start, stop;
  hiprandGenerator_t gen;
  std::random_device rd;
};

BENCHMARK_DEFINE_F(RandomArrayFixture, BM_RandomNumberGeneration)
(benchmark::State& state) {
  for (auto _ : state) {
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, rd()));
    CUDA_CALL(hipEventRecord(start));
    CURAND_CALL(hiprandGenerateUniform(gen, device_array, n));
    CUDA_CALL(hipEventRecord(stop));
    CURAND_CALL(hiprandDestroyGenerator(gen));
    hipEventSynchronize(stop);
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    state.SetIterationTime(static_cast<double>(milliseconds / 1.0e3));
  }
  state.SetItemsProcessed(static_cast<int64_t>(state.iterations()) *
                          state.range(0));
}

class ReductionFixture : public benchmark::Fixture {
 public:
  void SetUp(const ::benchmark::State& state) {
    n = state.range(0);
    array = new float[n];
    CUDA_CALL(hipMalloc(&device_array, n * sizeof(float)));
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, rd()));
  }

  void TearDown(const ::benchmark::State& state) {
    delete array;
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(device_array));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
  }

 protected:
  float* device_array;
  float* array;
  int n;
  hipEvent_t start, stop;
  hiprandGenerator_t gen;
  std::random_device rd;
};

BENCHMARK_DEFINE_F(ReductionFixture, BM_FindMax0)
(benchmark::State& state) {
  for (auto _ : state) {
    // Generate random number
    CURAND_CALL(hiprandGenerateUniform(gen, device_array, n));
    CUDA_CALL(hipMemcpy(array, device_array, n * sizeof(float),
                         hipMemcpyDeviceToHost));

    CUDA_CALL(hipEventRecord(start));
    FindMax0<<<64, 64, 64 * sizeof(float)>>>(device_array, n, device_array);
    CUDA_CALL(hipEventRecord(stop));
    FindMax0<<<1, 64, 64 * sizeof(float)>>>(device_array, n, device_array);

    float max_host = HostFindMax(array, n);
    CUDA_CALL(hipMemcpy(array, device_array, 1 * sizeof(float),
                         hipMemcpyDeviceToHost));
    EXPECT_FLOAT_EQ(max_host, array[0]);

    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    // Takes second
    state.SetIterationTime(milliseconds / 1.0e3);
  }
  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) *
                          state.range(0) * 4);
}

BENCHMARK_DEFINE_F(ReductionFixture, BM_FindMax1)
(benchmark::State& state) {
  for (auto _ : state) {
    // Generate random number
    CURAND_CALL(hiprandGenerateUniform(gen, device_array, n));
    CUDA_CALL(hipMemcpy(array, device_array, n * sizeof(float),
                         hipMemcpyDeviceToHost));

    CUDA_CALL(hipEventRecord(start));
    FindMax1<<<64, 64, 64 * sizeof(float)>>>(device_array, n, device_array);
    CUDA_CALL(hipEventRecord(stop));
    FindMax1<<<1, 64, 64 * sizeof(float)>>>(device_array, n, device_array);

    float max_host = HostFindMax(array, n);
    CUDA_CALL(hipMemcpy(array, device_array, 1 * sizeof(float),
                         hipMemcpyDeviceToHost));
    EXPECT_FLOAT_EQ(max_host, array[0]);

    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    // Takes second
    state.SetIterationTime(milliseconds / 1.0e3);
  }
  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) *
                          state.range(0) * 4);
}

// 2^(28+2) requires 2GB of VRAM
BENCHMARK_REGISTER_F(RandomArrayFixture, BM_RandomNumberGeneration)
    ->RangeMultiplier(2)
    ->Range(1 << 12, 1 << 28)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);

BENCHMARK_REGISTER_F(ReductionFixture, BM_FindMax0)
    ->RangeMultiplier(2)
    ->Range(1 << 12, 1 << 28)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);

BENCHMARK_REGISTER_F(ReductionFixture, BM_FindMax1)
    ->RangeMultiplier(2)
    ->Range(1 << 12, 1 << 28)
    ->UseManualTime()
    ->Unit(benchmark::kMillisecond);
