#include "hip/hip_runtime.h"
#include <glog/logging.h>

#include <random>
#include <iostream>

#include <hiprand.h>

// Compile with nvcc reduction.cu -lcurand -lglog -lgflags

// __global__ void RandomArray(float* array, int n) {

// }

__global__ void FindMax(const float* array, int n,
                        // output parameter
                        float* out_array) {
  // What happens if the shared memory is not enough? Runtime error?
  extern __shared__ float shared_array[];
  // First, for each thread of each block, look for max
  int thread_id = threadIdx.x;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  float local_max = array[id];
  // Find max for each thread
  for (int i = id + stride; i < n; i += stride) {
    if (local_max < array[i]) {
      local_max = array[i];
    }
  }

  // Each thread loads its max into each block's shared mem
  shared_array[thread_id] = local_max;
  __syncthreads();

  // Perform reduction upon each block
  for (int num_workers = blockDim.x / 2, step_size = 1; num_workers > 0;
       num_workers /= 2, step_size *= 2) {
    if (thread_id < num_workers) {
      int index = 2 * step_size * thread_id;
      float num1 = shared_array[index];
      float num2 = shared_array[index + step_size];

      if (num1 < num2)
        shared_array[index] = num2;
      else
        shared_array[index] = num1;
    }
    __syncthreads();
  }
  // All blocks have its max in shared_mem[0]
  // Write to global memory for synchronization
  if (thread_id == 0)
    out_array[blockIdx.x] = shared_array[0];
}

float HostFindMax(const float* array, int n) {
  if (n <= 0) return -1;
  float max = array[0];
  for (int i = 0; i < n; ++i) {
    if (array[i] > max) {
      max = array[i];
    }
  }
  return max;
}

#define CUDA_CALL(x)                                  \
  do {                                                \
    if ((x) != hipSuccess) {                         \
      printf("Error at %s:%d\n", __FILE__, __LINE__); \
      return EXIT_FAILURE;                            \
    }                                                 \
  } while (0)

#define CURAND_CALL(x)                                \
  do {                                                \
    if ((x) != HIPRAND_STATUS_SUCCESS) {               \
      printf("Error at %s:%d\n", __FILE__, __LINE__); \
      return EXIT_FAILURE;                            \
    }                                                 \
  } while (0)

int main(int argc, char* argv[]) {
  if (argc != 2) return -1;
  int n = std::atoi(argv[1]);
  float* device_array;
  float* array;
  hiprandGenerator_t gen;

  CUDA_CALL(hipMalloc(&device_array, n * sizeof(float)));
  array = new float[n];// malloc(n * sizeof(float));
  // max_index = new int; // malloc(max_index, sizeof(int));

  // Random number generation
  std::random_device rd;
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, rd()));
  CURAND_CALL(hiprandGenerateUniform(gen, device_array, n));
  CURAND_CALL(hiprandDestroyGenerator(gen));
  CUDA_CALL(hipDeviceSynchronize());

  // Since the FindMax would alter device_array, first copy to host
  CUDA_CALL(hipMemcpy(array, device_array, n * sizeof(float),
                       hipMemcpyDeviceToHost));
  // for (int i = 0; i < n-1; ++i)
  //   std::cout << array[i] << ' ';
  // std::cout << array[n-1] << '\n';
  float max_host = HostFindMax(array, n);

  // First FindMax() would assign the local maxima to the first 64 of device
  // array, which is the block size of the first call
  FindMax<<<64, 64, 64 * sizeof(float)>>>(device_array, n, device_array);
  FindMax<<<1, 64, 64 * sizeof(float)>>>(device_array, n, device_array);
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(array, device_array, 1 * sizeof(float),
                       hipMemcpyDeviceToHost));

  float max_device = array[0];
  CHECK_EQ(max_host, max_device);
  std::cout << max_host << '\n';
  // for (int i = 0; i < n-1; ++i)
  //   std::cout << array[i] << ' ';
  // std::cout << array[n-1] << '\n';

  // int kernel_max_index = *max_index;
  hipFree(device_array);
  //  hipFree(device_max_index);
  free(array);
  // free(max_index);
}